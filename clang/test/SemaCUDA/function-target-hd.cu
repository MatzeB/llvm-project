#include "hip/hip_runtime.h"
// facebook begin T63033708
// Test the Sema analysis of caller-callee relationships of host device
// functions when compiling CUDA code. There are 4 permutations of this test as
// host and device compilation are separate compilation passes, and clang has
// an option to allow host calls from host device functions. __CUDA_ARCH__ is
// defined when compiling for the device and TEST_WARN_HD when host calls are
// allowed from host device functions. So for example, if __CUDA_ARCH__ is
// defined and TEST_WARN_HD is not then device compilation is happening but
// host device functions are not allowed to call host functions.

// RUN: %clang_cc1 -fsyntax-only -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -verify %s
// RUN: %clang_cc1 -fsyntax-only -fcuda-allow-host-calls-from-host-device -verify %s -DTEST_WARN_HD -Werror
// RUN: %clang_cc1 -fsyntax-only -fcuda-is-device -fcuda-allow-host-calls-from-host-device -verify %s -DTEST_WARN_HD -Werror

#include "Inputs/hip/hip_runtime.h"

__host__ void hd1h(void);
#if defined(__CUDA_ARCH__) && !defined(TEST_WARN_HD)
// expected-note@-2 {{'hd1h' declared here}}
#endif
__device__ void hd1d(void);
#ifndef __CUDA_ARCH__
// expected-note@-2 {{'hd1d' declared here}}
#endif
__host__ void hd1hg(void);
__device__ void hd1dg(void);
#ifdef __CUDA_ARCH__
__host__ void hd1hig(void);
#if !defined(TEST_WARN_HD)
// expected-note@-2 {{'hd1hig' declared here}}
#endif
#else
__device__ void hd1dig(void); // expected-note {{'hd1dig' declared here}}
#endif
__host__ __device__ void hd1hd(void);
__global__ void hd1g(void);
#if defined(__CUDA_ARCH__)
// expected-note@-2 {{'hd1g' declared here}}
#endif

__host__ __device__ void hd1(void) {
#if defined(TEST_WARN_HD) && defined(__CUDA_ARCH__)
#endif
  hd1d();
#ifndef __CUDA_ARCH__
// expected-error@-2 {{reference to __device__ function 'hd1d' in __host__ __device__ function}}
#endif
  hd1h();
#if defined(__CUDA_ARCH__)
#if !defined(TEST_WARN_HD)
// expected-error@-3 {{reference to __host__ function 'hd1h' in __host__ __device__ function}}
#else
// expected-error@-5 {{calling __host__ function hd1h from __host__ __device__ function hd1 can lead to runtime errors}}
#endif
#endif

  // No errors as guarded
#ifdef __CUDA_ARCH__
  hd1d();
#else
  hd1h();
#endif

  // Errors as incorrectly guarded
#ifndef __CUDA_ARCH__
  hd1dig(); // expected-error {{reference to __device__ function 'hd1dig' in __host__ __device__ function}}
#else
  hd1hig();
#ifndef TEST_WARN_HD
// expected-error@-2 {{reference to __host__ function 'hd1hig' in __host__ __device__ function}}
#else
// expected-error@-4 {{calling __host__ function hd1hig from __host__ __device__ function hd1 can lead to runtime errors}}
#endif

#endif

  hd1hd();
  hd1g<<<1, 1>>>();
#ifdef __CUDA_ARCH__
  // expected-error@-2 {{reference to __global__ function 'hd1g' in __host__ __device__ function}}
#endif
}
// facebook end
